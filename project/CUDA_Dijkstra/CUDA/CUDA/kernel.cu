#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "AdjacencyMatrixGenerator.h"
#include <iostream>
#include <chrono>
#include <string>
#include <fstream>
#include <sstream>

// Funkcja do wczytywania macierzy sąsiedztwa z pliku tekstowegostd::vector<std::vector<std::vector<int>>> loadAdjacencyMatrixFromFile(const std::string& filename, int numberOfVertices) {
std::vector<std::vector<std::vector<int>>> loadAdjacencyMatrixFromFile(const std::string& filename, int numberOfVertices) {
    std::ifstream file(filename);
    std::vector<std::vector<std::vector<int>>> adjMatrix;

    if (file.is_open()) {
        adjMatrix.resize(numberOfVertices, std::vector<std::vector<int>>(numberOfVertices, std::vector<int>(numberOfVertices)));

        int value;
        std::string line;
        for (int k = 0; k < numberOfVertices; ++k) {
            for (int i = 0; i < numberOfVertices; ++i) {
                std::getline(file, line);
                std::istringstream iss(line);
                for (int j = 0; j < numberOfVertices; ++j) {
                    if (!(iss >> value)) {
                        std::cerr << "Błąd: Nie udało się wczytać wartości z pliku" << std::endl;
                        return adjMatrix;
                    }
                    adjMatrix[k][i][j] = value;
                }
            }
            // Pominięcie dodatkowej linii nowego wiersza
            std::getline(file, line);
            std::getline(file, line);
        }

        file.close();
    }
    else {
        std::cerr << "Błąd: Nie udało się otworzyć pliku" << std::endl;
    }

    return adjMatrix;
}



// Funkcja CUDA do wyznaczania najkrótszych ścieżek metodą Dijkstry
__global__
void dijkstraCUDA(const int* adjList, int numberOfVertices, int source, int* pathArray, bool* shortestPathSet) {
    // Identyfikator wątku
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Sprawdzenie czy wątek jest w granicach liczby wierzchołków
    if (tid >= numberOfVertices)
        return;

    // Inicjalizacja wartości najkrótszej ścieżki i zbioru odwiedzonych
    if (tid == source)
        pathArray[tid] = 0;
    else
        pathArray[tid] = INT_MAX;

    shortestPathSet[tid] = false;

    __syncthreads();

    for (int count = 0; count < numberOfVertices - 1; ++count) {
        // Znajdowanie wierzchołka o najmniejszej odległości
        int u;
        int minDistance = INT_MAX;
        for (int v = 0; v < numberOfVertices; ++v) {
            if (!shortestPathSet[v] && pathArray[v] <= minDistance) {
                minDistance = pathArray[v];
                u = v;
            }
        }

        // Oznaczenie wierzchołka jako odwiedzony
        shortestPathSet[u] = true;

        // Aktualizacja odległości dla sąsiednich wierzchołków
        for (int v = 0; v < numberOfVertices; ++v) {
            if (!shortestPathSet[v] && adjList[u * numberOfVertices + v] && pathArray[u] != INT_MAX &&
                pathArray[u] + adjList[u * numberOfVertices + v] < pathArray[v]) {
                pathArray[v] = pathArray[u] + adjList[u * numberOfVertices + v];
            }
        }

        __syncthreads();
    }
}


// Funkcja wywołująca algorytm Dijkstry na urządzeniu CUDA
int* dijkstraParallel(const int* adjList, int numEdges, int numberOfVertices, int source) {
    // Pomiar czasu wykonania
    auto startTime = std::chrono::high_resolution_clock::now();

    // Alokuje pamięć na wynikowe ścieżki i zbiór odwiedzonych wierzchołków na urządzeniu
    int* pathArray = new int[numberOfVertices];
    bool* shortestPathSet = new bool[numberOfVertices];
    int* devPathArray;
    bool* devVisited;
    int* devAdjList;

    hipMalloc(&devPathArray, sizeof(int) * numberOfVertices);
    hipMalloc(&devVisited, sizeof(bool) * numberOfVertices);
    hipMalloc(&devAdjList, sizeof(int) * numberOfVertices * numberOfVertices);

    // Inicjalizacja odległości i zbioru odwiedzonych wierzchołków na hostingu
    for (int i = 0; i < numberOfVertices; ++i) {
        pathArray[i] = INT_MAX;
        shortestPathSet[i] = false;
    }
    pathArray[source] = 0;

    // Kopiowanie danych z hosta do urządzenia
    hipMemcpy(devPathArray, pathArray, numberOfVertices * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(devVisited, shortestPathSet, numberOfVertices * sizeof(bool), hipMemcpyHostToDevice);
    hipMemcpy(devAdjList, adjList, numberOfVertices * numberOfVertices * sizeof(int), hipMemcpyHostToDevice);

    // Wywołanie funkcji dijkstraCUDA na urządzeniu
    dijkstraCUDA << <(numberOfVertices + 255) / 256, 256 >> > (devAdjList, numberOfVertices, source, devPathArray, devVisited);

    // Kopiowanie wyników z urządzenia do hosta
    hipMemcpy(pathArray, devPathArray, numberOfVertices * sizeof(int), hipMemcpyDeviceToHost);


    // Pomiar czasu wykonania
    auto endTime = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(endTime - startTime).count();

    // Synchronizacja urządzenia
    hipDeviceSynchronize();

    

    // Wyświetlanie czasu wykonania
    std::cout << "Execution time for CUDA Dijkstra algorithm: " << duration << " microseconds" << std::endl;

    // Zwolnienie pamięci na urządzeniu i hostingu
    hipFree(devPathArray);
    hipFree(devVisited);
    hipFree(devAdjList);
    delete[] shortestPathSet;

    return pathArray; // Zwraca wynikowe ścieżki
}


void PrintMinPath(int* pathArray, int numberOfVertices, int source) {
    std::cout << "Odleglosc od wierzcholka " << source << " do innych wierzcholkow:" << std::endl;

    for (int i = 0; i < numberOfVertices; ++i) {
        if (pathArray[i] == INT_MAX || pathArray[i] == 100000) {
            std::cout << "Unknown ";
        }
        else {
            std::cout << pathArray[i] << " ";
        }
    }
    std::cout << std::endl;

    delete[] pathArray;
}


int main()
{
    for (int i = 100; i < 1000; i = i + 100) {

        int numberOfVertices = i;
        const int adjListSize = numberOfVertices * numberOfVertices * numberOfVertices;
        int* adjList = new int[adjListSize];


        // Generowanie macierzy sąsiedztwa
        AdjacencyMatrixGenerator adj;
        std::vector<std::vector<std::vector<int>>> adjListWrapper = adj.generateGraph(numberOfVertices);

        // Wczytanie macierzy sąsiedztwa z pliku
        //std::string filename = "test_adj_matrix.txt";
        //std::vector<std::vector<std::vector<int>>> adjListWrapper = loadAdjacencyMatrixFromFile(filename, numberOfVertices);


        // Dostosowanie do listy jednowymiarowej
        for (int k = 0; k < numberOfVertices; ++k) {
            for (int i = 0; i < numberOfVertices; ++i) {
                for (int j = 0; j < numberOfVertices; ++j) {
                    adjList[k * numberOfVertices * numberOfVertices + i * numberOfVertices + j] = adjListWrapper[k][i][j];
                }
            }
        }

        std::cout << numberOfVertices << " vertices" << std::endl;
        // Wywołanie algorytmu Dijkstry na GPU
        printf("Results from GPU Dijkstra:\n");
        int* pathArray = dijkstraParallel(adjList, adjListSize, numberOfVertices, 0);
        //PrintMinPath(pathArray, numberOfVertices, 0);

        delete[] adjList;
    }
    return 0;
}
